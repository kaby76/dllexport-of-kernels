
#include "hip/hip_runtime.h"


#include <stdio.h>

__declspec(dllexport) __global__ void doKernel2(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = c[i] + b[i] + a[i];
}

// Can declare, but cannot link in "hw".
__declspec(dllexport) __device__ void fun(int *c, const int *a, const int *b)
{
	c[0] = c[0] + b[0] + a[0];
}

__global__ void doKernel3(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = c[i] + b[i] + a[i];
}
