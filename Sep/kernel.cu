
#include "hip/hip_runtime.h"


#include <stdio.h>

__declspec(dllexport) __global__ void doKernel2(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = c[i] + b[i] + a[i];
}

