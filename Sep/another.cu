
#include "hip/hip_runtime.h"


#include <stdio.h>


__global__ void doKernel4(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = c[i] + b[i] + a[i];
}

__device__ void funny()
{

}