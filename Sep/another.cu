
#include "hip/hip_runtime.h"


#include <stdio.h>

__device__ void fun(int *c, const int *a, const int *b);

__global__ void doKernel4(int *c, const int *a, const int *b)
{
	fun(c, a, b);
	int i = threadIdx.x;
	c[i] = c[i] + b[i] + a[i];
}
